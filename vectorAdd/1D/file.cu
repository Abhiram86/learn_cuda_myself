#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void addVectors1D(int* a, int *b, int *c, int n) {
    int i = threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

int main() {
    int *a, *b, *c, n = 6;
    a = (int*)malloc(sizeof(int) * n);
    b = (int*)malloc(sizeof(int) * n);
    c = (int*)malloc(sizeof(int) * n);

    for (int i = 0; i < n; i++) {
        printf("Enter value for a[%d]: ", i);
        scanf("%d", &a[i]);
    }
    
    for (int i = 0; i < n; i++) {
        printf("Enter value for b[%d]: ", i);
        scanf("%d", &b[i]);
    }
    
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, sizeof(int) * n);
    hipMalloc((void**)&d_b, sizeof(int) * n);
    hipMalloc((void**)&d_c, sizeof(int) * n);

    hipMemcpy(d_a, a, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int) * n, hipMemcpyHostToDevice);

    addVectors1D<<<1, n>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, sizeof(int) * n, hipMemcpyDeviceToHost);

    printf("\n");
    
    for (int i = 0; i < n; i++) {
        printf("%d ", c[i]);
    }

    free(a);
    free(b);
    free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}