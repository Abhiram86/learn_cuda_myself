#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../include/Matrix.h"

__global__ void addVectors2D(int *a, int *b, int *c, int n) {
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    if (row < n && col < n) {
        int idx = row * n + col;
        c[idx] = a[idx] + b[idx];
    }
}

int main(){
    Matrix a(3, 3), b(3, 3), c(3, 3);

    const int matSize = a.cols * a.rows * sizeof(int);

    a.MatGetInput();
    printf("\n");
    b.MatGetInput();

    int *flat_a = a.FlattenCopy();
    int *flat_b = b.FlattenCopy();
    int *flat_c = c.FlattenCopy();

    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, matSize);
    hipMalloc((void**)&d_b, matSize);
    hipMalloc((void**)&d_c, matSize);

    hipMemcpy(d_a, flat_a, matSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, flat_b, matSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((a.cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
    (a.rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    addVectors2D<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, a.cols);


    hipMemcpy(flat_c, d_c, matSize, hipMemcpyDeviceToHost);

    c.FromFLatArray(flat_c);

    printf("a matrix\n");
    a.PrintMatrix();
    printf("b matrix\n");
    b.PrintMatrix();
    printf("c matrix\n");
    c.PrintMatrix();

    free(flat_a);
    free(flat_b);
    free(flat_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}