#include<hip/hip_runtime.h>
#include<stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        printf("Device #%d: %s\n", i, prop.name);
        printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("  Total global memory: %lu bytes\n", prop.totalGlobalMem);
        printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("  Max threads per dimension (x,y,z): %d, %d, %d\n",
               prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Max grid size (x,y,z): %d, %d, %d\n",
               prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("  Number of multiprocessors: %d\n", prop.multiProcessorCount);
        printf("  Warp size: %d\n", prop.warpSize);
        printf("\n");
    }

    printf("hello, world!\n");
}